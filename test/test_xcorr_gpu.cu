#include "hip/hip_runtime.h"
/**
 *  Test calculus of cross correlation
 */

#include "../modules/CPP-test-unit/tester.hh"
#include "../modules/CPP-lockfree-queue/fixed_size_lockfree_queue.hh"

#include "../src/chunk.hh"
#include "../src/numeric_parser.hh"
#include "../src/cuda_numeric_consumer.hh"

#include <stdexcept>
#include <string>
#include <vector>
#include <memory>
#include <random>
#include <valarray>
#include <iostream>
#include <algorithm>

template <typename T>
std::valarray<T> evaluate(const std::vector<std::vector<T>>& dataset) {
    auto columns = dataset.size();
    auto couples = columns*(columns-1)/2;
    std::valarray<T> ans(couples);
    int pairIdx{};
    for (int i{}; i+1!=columns; ++i) {
        for (int j{i+1}; j!=columns; ++j) {
            ans[pairIdx++] += math::statistics::pearson_correlation_coefficient(dataset[i], dataset[j]).compute();
        }
    }
    return ans;
}


/**
 * @brief Test if the explicit calculus give the same result of the
 * one performed by the class numeric_consumer
 * 
 * @return tester 
 */
tester test_xcorr([](){
    using test_type = double;
    constexpr std::size_t rows = 1000;
    constexpr std::size_t cols = 30;

    // initialize random number generator
    std::default_random_engine generator;
    std::uniform_real_distribution<test_type> distribution(30,77);

    // generate matrix
    std::vector<std::vector<test_type>> dataset; dataset.reserve(cols);
    for (int c{}; c!=cols; ++c) {
        std::vector<test_type> column; column.reserve(rows);
        for (int r{}; r!=rows; ++r) {
            column.push_back(distribution(generator));
        }
        dataset.push_back(std::move(column));
    }

    // generate queue:
    //  INPUT queue with chunk to be parsed
    auto chunkQueue = std::shared_ptr<lockfree_queue::fixed_size_lockfree_queue<chunk<test_type>>>(
        new lockfree_queue::fixed_size_lockfree_queue<chunk<test_type>>(1)
    );

    // generate chunk
    chunk<test_type> cnk(rows, cols);
    for (int r{}; r!=rows; ++r) {
        for (int c{}; c!=cols; ++c) {
            cnk.push_back(dataset[c][r]);
            if (cnk.at(r,c) != dataset[c][r]) {
                throw std::logic_error("Strange chunk insertion.");
            }
        }
    }

    // push chunk in queue
    auto cnk_ptr = std::make_unique<decltype(cnk)>(std::move(cnk));
    if (!chunkQueue->offer(cnk_ptr)) {
        throw std::logic_error("Failed chunk insertion.");
    }

    cuda_numeric_consumer<test_type> consumer(cols, chunkQueue);

    // analyze results
    consumer.analyze();

    auto res = consumer.get_results_and_invalidate();
    std::valarray<test_type> casted(res.size());
    std::transform(std::begin(res), std::end(res), std::begin(casted),
        [](decltype(res[0]) it){
            return it.compute();
        });

    auto evaluated = evaluate(dataset);
    // check equivalence
    if ((casted != evaluated).min() == false) {
        std::cerr << "casted | evaluated:\n";
        int idx{};
        for (int i{}; i+1!=cols; ++i) {
            for (int j{i+1}; j!=cols; ++j) {
                using namespace std::literals;
                std::cerr << " (" << i << "," << j << ") " <<
                    casted[idx] << " | " << evaluated[idx] << '\n';
            }
        }

        throw std::logic_error("Failed analysis.");
    }

});